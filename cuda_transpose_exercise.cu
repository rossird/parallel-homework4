#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"
//#include "utils.h"

const int N= 1024;  // matrix size will be NxN

int compare_matrices(float *gpu, float *ref, int N)
{
  int result = 0;
  for(int j=0; j < N; j++)
    for(int i=0; i < N; i++)
      if (ref[i + j*N] != gpu[i + j*N])
        {//printf("i: %f\tj: %f\tref: %i\tgpu: %i\n",i, j, ref[i + j*N], gpu[i + j*N]);
        result = 1;}
  return result;
}


// fill a matrix with sequential numbers in the range 0..N-1
void fill_matrix(float *mat, int N)
{
  for(int j=0; j < N * N; j++)
    mat[j] = (float) j;
}

// The following functions and kernels are for your references
void 
transpose_CPU(float in[], float out[])
{
  for(int j=0; j < N; j++)
    for(int i=0; i < N; i++)
      out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched on a single thread
__global__ void 
transpose_serial(float in[], float out[])
{
  for(int j=0; j < N; j++)
    for(int i=0; i < N; i++)
      out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched with one thread per row of output matrix
__global__ void 
transpose_parallel_per_row(float in[], float out[])
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  for(int j=0; j < N; j++)
    out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}



// Write two tiled versions of transpose -- One using shared memory. 
// To be launched with one thread per element, in KxK threadblocks.
// You will determine for each thread (x,y) in tile the element (i,j) of global output matrix. 

__global__ void 
transpose_parallel_per_element_tiled(float in[], float out[])
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  //Ignore any thread mapped to element outside the matrix
  if (x >= N || y >= N)
    return;
    
  //Swap  two elements.
  out[x + y*N] = in[y + x*N]; // out(j,i) = in(i,j)

}

__global__ void 
transpose_parallel_per_element_tiled_shared(float in[], float out[])
{
  extern __shared__ float tileData[];
  
  int x =  blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  if (x >= N || y >= N)
    return;
  
  int blockStart = N * blockIdx.y * blockDim.y + blockIdx.x * blockDim.x;
  
  //First thread will copy the data into share memory
  if(threadIdx.x == 0 && threadIdx.y == 0) {
    for(int x = 0; x < blockDim.x; x++) {
      for(int y = 0; y < blockDim.y; y++) {
        tileData[x+blockDim.x*y] = in[blockStart + x + N * y];
      }
    }
  }
  __syncthreads();
  
  out[y + x*N] = tileData[threadIdx.x + threadIdx.y * blockDim.x];
  
}

int main(int argc, char **argv)
{
  int numbytes = N * N * sizeof(float);

  float *in = (float *) malloc(numbytes);
  float *out = (float *) malloc(numbytes);
  float *gold = (float *) malloc(numbytes);

  fill_matrix(in, N);
  transpose_CPU(in, gold);

  float *d_in, *d_out;

  hipMalloc(&d_in, numbytes);
  hipMalloc(&d_out, numbytes);
  hipMemcpy(d_in, in, numbytes, hipMemcpyHostToDevice);

  GpuTimer timer;


  timer.Start();
  transpose_serial<<<1,1>>>(d_in, d_out);
  timer.Stop();
  for (int i=0; i < N*N; ++i){out[i] = 0.0;}
  hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
  printf("transpose_serial: %g ms.\nVerifying ...%s\n", 
      timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");

   
  hipMemcpy(d_out, d_in, numbytes, hipMemcpyDeviceToDevice); //clean d_out
  timer.Start();
  transpose_parallel_per_row<<<1,N>>>(d_in, d_out);
  timer.Stop();
  for (int i=0; i < N*N; ++i){out[i] = 0.0;}  //clean out
  hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
  printf("transpose_parallel_per_row: %g ms.\nVerifying ...%s\n", 
        timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");

  hipMemcpy(d_out, d_in, numbytes, hipMemcpyDeviceToDevice); //clean d_out
  
  // Tiled versions
  int K = 20;
  if(argc > 1) {
    K = atoi(argv[1]);
  }
  int gridWidth = ceil(1.0 * N / K);
  
  dim3 blocks_tiled(gridWidth,gridWidth);
  dim3 threads_tiled(K,K);
  timer.Start();
  transpose_parallel_per_element_tiled<<<blocks_tiled,threads_tiled>>>(d_in, d_out);
  timer.Stop();
  for (int i=0; i < N*N; ++i){out[i] = 0.0;}
  hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
  printf("transpose_parallel_per_element_tiled %dx%d: %g ms.\nVerifying ...%s\n", 
       K, K, timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");
       
  hipMemcpy(d_out, d_in, numbytes, hipMemcpyDeviceToDevice); //clean d_out
  
  dim3 blocks_tiled_sh(gridWidth,gridWidth);
  dim3 threads_tiled_sh(K,K);
  size_t sharedMemSize = K * K * sizeof(float);
  timer.Start();
  transpose_parallel_per_element_tiled_shared<<<blocks_tiled_sh,threads_tiled_sh, sharedMemSize>>>(d_in, d_out);
  timer.Stop();
  for (int i=0; i < N*N; ++i){out[i] = 0.0;}
  hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
  printf("transpose_parallel_per_element_tiled_shared %dx%d: %g ms.\nVerifying ...%s\n", 
       K, K, timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");

  hipFree(d_in);
  hipFree(d_out);
}
